#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <algorithm>
#include <cmath>

using namespace std;
#include "AudioFile.h"

#define PROJECT_BINARY_DIR "C:/Users/mlaramie24/source/repos/AudioWork/AudioWork"

// Returns an AudioFile object, also fills a given vector with the 1d conversion of the audio file.
AudioFile<float> loadAudio(vector<float>& v) {
    //---------------------------------------------------------------
    // 1. Set a file path to the audio file

    const std::string inputFilePath = std::string(PROJECT_BINARY_DIR) + "/sample_3.wav";

    //---------------------------------------------------------------
    // 2. Load the audio file

    AudioFile<float> a;
    bool loadedOK = a.load(inputFilePath);

    printf("%d %d\n", a.getNumChannels(), a.getNumSamplesPerChannel());

    //---------------------------------------------------------------
    // Convert data from Multi-Dimensional Array -> One-Dimensional Array
    for (int i = 0; i < a.getNumSamplesPerChannel(); i++) {
        v.push_back((a.samples[0][i] + a.samples[1][i]) / 2);
    }
    return a;
}

void findBins(int index, hipfftComplex* hod, hipfftComplex* crazy, int size) {
    for (int i = 0; i < size / 2 + 1; i++) {
        if (i > index - 100 && i < index + 100) {
            crazy[i].x = hod[i].x;
            crazy[i].y = hod[i].y;
        }
        else {
            crazy[i].x = 0.0f;
            crazy[i].y = 0.0f;
        }
    }
    for (int i = index - 100; i < index + 100; i++) {
        hod[i].x = 0.0f;
        hod[i].y = 0.0f;
    }
    cout << "INDEX: " << index << endl;
}

int main() {
    std::cout << "**********************" << std::endl;
    std::cout << "cuFFT audio transform" << std::endl;
    std::cout << "**********************" << std::endl << std::endl;


    //---------------------------------------------------------------
    // Step 1: Gather Input Data
    vector<float> audio;
    AudioFile<float> a = loadAudio(audio);
    int size = audio.size();

    hipfftReal* hostInputData = (hipfftReal*)malloc(size * sizeof(hipfftReal));
    for (int i = 0; i < size; i++) {
        hostInputData[i] = (hipfftReal)audio[i];
    }

    cout << "REAL DATA BT: " << endl;
    for (int i = 0; i < 30; i++) {
        printf("%i %f\n", i, hostInputData[i]);
    }

    //---------------------------------------------------------------
    // Step 2: Device memory allocation
    hipfftReal* deviceInputData;
    hipMalloc((void**)&deviceInputData, size * sizeof(hipfftReal));
    hipMemcpy(deviceInputData, hostInputData, size * sizeof(hipfftReal), hipMemcpyHostToDevice);

    //---------------------------------------------------------------
    // Step 3: Allocate host & device memory for output data collection
    hipfftComplex* hostOutputData = (hipfftComplex*)malloc((size / 2 + 1) * 1 * sizeof(hipfftComplex));

    hipfftComplex* deviceOutputData;
    hipMalloc((void**)&deviceOutputData, (size / 2 + 1) * sizeof(hipfftComplex));

    // Setup cufft plan and handle
    hipfftHandle handle;
    hipfftResult cufftStatus = hipfftPlan1d(&handle, size, HIPFFT_R2C, 1);

    //---------------------------------------------------------------
    // Step 4: Execute R2C FFT
    cufftStatus = hipfftExecR2C(handle, (hipfftReal*)deviceInputData, deviceOutputData);

    //---------------------------------------------------------------
    // Step 5: Transfer results from Device -> Host
    hipMemcpy(hostOutputData, deviceOutputData, (size / 2 + 1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    //---------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
    // NEXT PART: C2R 1

    vector<float> vecto;
    // Data Manip
    for (int i = 0; i < size / 2 + 1; i++) {
        //printf("%f %f\n", hostOutputData[i].x, hostOutputData[i].y);
        float mag = sqrt((hostOutputData[i].x * hostOutputData[i].x) + (hostOutputData[i].y * hostOutputData[i].y));
        vecto.push_back(mag);
    }

    // Parse data into segments based on estimated minimum magnitude for a point of interest

    hipfftComplex* crazee = (hipfftComplex*)malloc((size / 2 + 1) * sizeof(hipfftComplex));
    for (int i = 0; i < size / 2 + 1; i++) {
        crazee[i].x = hostOutputData[i].x;
        crazee[i].y = hostOutputData[i].y;
    }

    float highest = *max_element(vecto.begin(), vecto.end());
    //float lowest = *min_element(vecto.begin(), vecto.end());

    // If you want to generate a CSV file 
    std::ofstream ffout("jingle_out.csv");
    ffout << "\"Re\"" << "," << "\"Im\"" << std::endl;
    for (int i = 0; i < (size / 2 + 1); i++)
    {
        ffout << hostOutputData[i].x << "," << hostOutputData[i].y << std::endl;
    }
    ffout.close();

    for (int i = 0; i < size / 2 + 1; i++) {
        if (vecto[i] == highest) {
            findBins(i, hostOutputData, crazee, size);
        }
    }

    //---------------------------------------------------------------
    // Step 1: Gather Input Data
    hipfftComplex* hostInputData2 = (hipfftComplex*)malloc((size / 2 + 1) * sizeof(hipfftComplex));
    for (int i = 0; i < size / 2 + 1; i++) {
        hostInputData2[i].x = hostOutputData[i].x;
        hostInputData2[i].y = hostOutputData[i].y;
    }

    // Step 2: Device memory Allocation
    hipfftComplex* deviceInputData2;
    hipMalloc((void**)&deviceInputData2, (size / 2 + 1) * sizeof(hipfftComplex));
    hipMemcpy(deviceInputData2, hostInputData2, (size / 2 + 1) * sizeof(hipfftComplex), hipMemcpyHostToDevice);

    //---------------------------------------------------------------
    // Step 3: Allocate host & device memory for output data collection
    hipfftReal* hostOutputData2 = (hipfftReal*)malloc(size * sizeof(hipfftReal));
    hipfftReal* deviceOutputData2;
    hipMalloc((void**)&deviceOutputData2, size * sizeof(hipfftReal));

    // Setup cufft plan and handle
    hipfftHandle handle2;
    hipfftResult cufftStatus2 = hipfftPlan1d(&handle2, (size / 2 + 1), HIPFFT_C2R, 1);

    //---------------------------------------------------------------
    // Step 4: Execute C2R FFT
    cufftStatus2 = hipfftExecC2R(handle2, (hipfftComplex*)deviceInputData2, deviceOutputData2);

    //---------------------------------------------------------------
    // Step 5: Transfer results from Device -> Host
    hipMemcpy(hostOutputData2, deviceOutputData2, size * sizeof(hipfftReal), hipMemcpyDeviceToHost);

    //--------------------------------------------------------------------------------------------------------------------------------
    // NEXT PART: C2R 2

    // Step 1: Gather input Data

    hipfftComplex* hostInputData3 = (hipfftComplex*)malloc((size / 2 + 1) * sizeof(hipfftComplex));
    for (int i = 0; i < size / 2 + 1; i++) {
        hostInputData3[i].x = crazee[i].x;
        hostInputData3[i].y = crazee[i].y;
    }


    // Step 2: Device memory Allocation
    hipfftComplex* deviceInputData3;
    hipMalloc((void**)&deviceInputData3, (size / 2 + 1) * sizeof(hipfftComplex));
    hipMemcpy(deviceInputData3, hostInputData3, (size / 2 + 1) * sizeof(hipfftComplex), hipMemcpyHostToDevice);

    //---------------------------------------------------------------
    // Step 3: Allocate host & device memory for output data collection
    hipfftReal* hostOutputData3 = (hipfftReal*)malloc(size * sizeof(hipfftReal));
    hipfftReal* deviceOutputData3;
    hipMalloc((void**)&deviceOutputData3, size * sizeof(hipfftReal));

    // Setup cufft plan and handle
    hipfftHandle handle3;
    hipfftResult cufftStatus3 = hipfftPlan1d(&handle3, (size / 2 + 1), HIPFFT_C2R, 1);

    //---------------------------------------------------------------
    // Step 4: Execute C2R FFT
    cufftStatus3 = hipfftExecC2R(handle3, (hipfftComplex*)deviceInputData3, deviceOutputData3);

    //---------------------------------------------------------------
    // Step 5: Transfer results from Device -> Host
    hipMemcpy(hostOutputData3, deviceOutputData3, size * sizeof(hipfftReal), hipMemcpyDeviceToHost);









    // Cleanup
    hipfftDestroy(handle);
    hipfftDestroy(handle2);
    hipfftDestroy(handle3);



    /*cout << "REAL DATA AT: " << endl;
    for (int j = 0; j < parses[i].size(); j++) {
        printf("%i %f\n", j, parses[i][j]);
    }*/


    AudioFile<float> output;
    string outputFilePath = "result_audio2.wav"; // change this to somewhere useful for you
    cout << outputFilePath << endl;

    for (int j = 0; j < size; j++) {
        output.samples[0].push_back(hostOutputData2[j] / size);
        output.samples[0].push_back(0);
    }
    output.save(outputFilePath, AudioFileFormat::Wave);
        cout << audio.size() << endl;

        AudioFile<float> output2;
        string outputFilePath2 = "result_audio3.wav"; // change this to somewhere useful for you
        cout << outputFilePath2 << endl;

        for (int j = 0; j < size; j++) {
            output2.samples[0].push_back(hostOutputData3[j] / size);
            output2.samples[0].push_back(0);
        }
        output2.save(outputFilePath2, AudioFileFormat::Wave);
        cout << audio.size() << endl;
    


}