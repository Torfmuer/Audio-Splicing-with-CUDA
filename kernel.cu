#include "hip/hip_runtime.h"
//---------------------------------------------------------------
// AUDIO SLICING FINAL PROJECT
// CS 315 Distributed Scalable Computing
// Dr. Qian Mao 
// Whitworth University
// DEVELOPED BY:
// LYDIA CALDERON-ACEITUNO, MICHAEL LARAMIE, OWEN FOSTER
//---------------------------------------------------------------
// USING LIBRARY DEVELOPED BY ADAM STARK
// https://github.com/adamstark/AudioFile
// https://www.adamstark.co.uk
//---------------------------------------------------------------

#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <algorithm>
#include <cmath>

using namespace std;
#include "AudioFile.h"

#define PROJECT_BINARY_DIR "C:/Users/mlaramie24/source/repos/AudioWork/AudioWork"

// Returns an AudioFile object, also fills a given vector with the 1d conversion of the audio file.
AudioFile<float> loadAudio(vector<float>& v) {
    //---------------------------------------------------------------
    // 1. Set a file path to the audio file

    const std::string inputFilePath = std::string(PROJECT_BINARY_DIR) + "/sample_3.wav";

    //---------------------------------------------------------------
    // 2. Load the audio file

    AudioFile<float> a;
    bool loadedOK = a.load(inputFilePath);

    printf("%d %d\n", a.getNumChannels(), a.getNumSamplesPerChannel()); // print stats for testing

    //---------------------------------------------------------------
    // Convert data from Multi-Dimensional Array -> One-Dimensional Array
    for (int i = 0; i < a.getNumSamplesPerChannel(); i++) {
        v.push_back((a.samples[0][i] + a.samples[1][i]) / 2);
    }
    return a;
}

// Function to slice the complex data into two lists
// Takes in the index of the highest magnitude frequency
// Slices 100 indices below and above
void findBins(int index, hipfftComplex* hod, hipfftComplex* crazy, int size) {
    // Grab all complex data points other than the slice
    for (int i = 0; i < size / 2 + 1; i++) {
        if (i > index - 100 && i < index + 100) {
            crazy[i].x = hod[i].x;
            crazy[i].y = hod[i].y;
        }
        else {
            crazy[i].x = 0.0f;
            crazy[i].y = 0.0f;
        }
    }
    // Grab the slice
    for (int i = index - 100; i < index + 100; i++) {
        hod[i].x = 0.0f;
        hod[i].y = 0.0f;
    }
    cout << "INDEX: " << index << endl; // print index for testing
}

//---------------------------------------------------------------
// Main
int main() {
    // Print basic info for testing in the console
    std::cout << "**********************" << std::endl;
    std::cout << "cuFFT audio transform" << std::endl;
    std::cout << "**********************" << std::endl << std::endl;


    //---------------------------------------------------------------
    // Step 1: Gather Input Data
    vector<float> audio;
    AudioFile<float> a = loadAudio(audio);
    int size = audio.size();

    // Put all data points from audio file into a hipfftReal* object 
    // to use in FFT
    hipfftReal* hostInputData = (hipfftReal*)malloc(size * sizeof(hipfftReal));
    for (int i = 0; i < size; i++) {
        hostInputData[i] = (hipfftReal)audio[i];
    }

    // Print out sample data to test
    cout << "REAL DATA BT: " << endl;
    for (int i = 0; i < 30; i++) {
        printf("%i %f\n", i, hostInputData[i]);
    }

    //---------------------------------------------------------------
    // Step 2: Device memory allocation
    hipfftReal* deviceInputData;
    hipMalloc((void**)&deviceInputData, size * sizeof(hipfftReal));
    hipMemcpy(deviceInputData, hostInputData, size * sizeof(hipfftReal), hipMemcpyHostToDevice);

    //---------------------------------------------------------------
    // Step 3: Allocate host & device memory for output data collection
    hipfftComplex* hostOutputData = (hipfftComplex*)malloc((size / 2 + 1) * 1 * sizeof(hipfftComplex));

    hipfftComplex* deviceOutputData;
    hipMalloc((void**)&deviceOutputData, (size / 2 + 1) * sizeof(hipfftComplex));

    // Setup cufft plan and handle
    hipfftHandle handle;
    hipfftResult cufftStatus = hipfftPlan1d(&handle, size, HIPFFT_R2C, 1);

    //---------------------------------------------------------------
    // Step 4: Execute R2C FFT
    cufftStatus = hipfftExecR2C(handle, (hipfftReal*)deviceInputData, deviceOutputData);

    //---------------------------------------------------------------
    // Step 5: Transfer results from Device -> Host
    hipMemcpy(hostOutputData, deviceOutputData, (size / 2 + 1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    
    //---------------------------------------------------------------
    // Create vector of magnitudes of each complex data point
    // vector has length of (size / 2) + 1
    vector<float> vecto;
    for (int i = 0; i < size / 2 + 1; i++) {
        //printf("%f %f\n", hostOutputData[i].x, hostOutputData[i].y);
        float mag = sqrt((hostOutputData[i].x * hostOutputData[i].x) + (hostOutputData[i].y * hostOutputData[i].y));
        vecto.push_back(mag);
    }

    // Clone complex data in order to have two copies to slice in
    hipfftComplex* crazee = (hipfftComplex*)malloc((size / 2 + 1) * sizeof(hipfftComplex));
    for (int i = 0; i < size / 2 + 1; i++) {
        crazee[i].x = hostOutputData[i].x;
        crazee[i].y = hostOutputData[i].y;
    }

    // Find the index with the highest magnitude
    // this is equivalent to the loudest frequency
    float highest = *max_element(vecto.begin(), vecto.end());

    // If you want to generate a CSV file
    // CSV file needed for graphing function 
    std::ofstream ffout("jingle_out.csv");
    ffout << "\"Re\"" << "," << "\"Im\"" << std::endl;
    for (int i = 0; i < (size / 2 + 1); i++)
    {
        ffout << hostOutputData[i].x << "," << hostOutputData[i].y << std::endl;
    }
    ffout.close();

    //---------------------------------------------------------------
    // Iterate through vector of magnitudes to find the index of the highest magnitude
    // once highest magnitude index is found, call findBins(...) to slice the data around the highest index
    for (int i = 0; i < size / 2 + 1; i++) {
        if (vecto[i] == highest) {
            findBins(i, hostOutputData, crazee, size);
        }
    }

    //---------------------------------------------------------------
    // Begin complex to real inverse fourier transform to convert the now sliced complex data into 
    // real data to write to audio file
    //---------------------------------------------------------------
    // Surrounding data conversion back to real data
    // Step 1: Gather Input Data
    hipfftComplex* hostInputData2 = (hipfftComplex*)malloc((size / 2 + 1) * sizeof(hipfftComplex));
    for (int i = 0; i < size / 2 + 1; i++) {
        hostInputData2[i].x = hostOutputData[i].x;
        hostInputData2[i].y = hostOutputData[i].y;
    }

    // Step 2: Device memory Allocation
    hipfftComplex* deviceInputData2;
    hipMalloc((void**)&deviceInputData2, (size / 2 + 1) * sizeof(hipfftComplex));
    hipMemcpy(deviceInputData2, hostInputData2, (size / 2 + 1) * sizeof(hipfftComplex), hipMemcpyHostToDevice);

    //---------------------------------------------------------------
    // Step 3: Allocate host & device memory for output data collection
    hipfftReal* hostOutputData2 = (hipfftReal*)malloc(size * sizeof(hipfftReal));
    hipfftReal* deviceOutputData2;
    hipMalloc((void**)&deviceOutputData2, size * sizeof(hipfftReal));

    // Setup cufft plan and handle
    hipfftHandle handle2;
    hipfftResult cufftStatus2 = hipfftPlan1d(&handle2, (size / 2 + 1), HIPFFT_C2R, 1);

    //---------------------------------------------------------------
    // Step 4: Execute C2R FFT
    cufftStatus2 = hipfftExecC2R(handle2, (hipfftComplex*)deviceInputData2, deviceOutputData2);

    //---------------------------------------------------------------
    // Step 5: Transfer results from Device -> Host
    hipMemcpy(hostOutputData2, deviceOutputData2, size * sizeof(hipfftReal), hipMemcpyDeviceToHost);

    //---------------------------------------------------------------
    // Sliced data conversion back to real data
    // Step 1: Gather input Data
    hipfftComplex* hostInputData3 = (hipfftComplex*)malloc((size / 2 + 1) * sizeof(hipfftComplex));
    for (int i = 0; i < size / 2 + 1; i++) {
        hostInputData3[i].x = crazee[i].x;
        hostInputData3[i].y = crazee[i].y;
    }

    // Step 2: Device memory Allocation
    hipfftComplex* deviceInputData3;
    hipMalloc((void**)&deviceInputData3, (size / 2 + 1) * sizeof(hipfftComplex));
    hipMemcpy(deviceInputData3, hostInputData3, (size / 2 + 1) * sizeof(hipfftComplex), hipMemcpyHostToDevice);

    //---------------------------------------------------------------
    // Step 3: Allocate host & device memory for output data collection
    hipfftReal* hostOutputData3 = (hipfftReal*)malloc(size * sizeof(hipfftReal));
    hipfftReal* deviceOutputData3;
    hipMalloc((void**)&deviceOutputData3, size * sizeof(hipfftReal));

    // Setup cufft plan and handle
    hipfftHandle handle3;
    hipfftResult cufftStatus3 = hipfftPlan1d(&handle3, (size / 2 + 1), HIPFFT_C2R, 1);

    //---------------------------------------------------------------
    // Step 4: Execute C2R FFT
    cufftStatus3 = hipfftExecC2R(handle3, (hipfftComplex*)deviceInputData3, deviceOutputData3);

    //---------------------------------------------------------------
    // Step 5: Transfer results from Device -> Host
    hipMemcpy(hostOutputData3, deviceOutputData3, size * sizeof(hipfftReal), hipMemcpyDeviceToHost);

    //---------------------------------------------------------------
    // Cleanup
    hipfftDestroy(handle);
    hipfftDestroy(handle2);
    hipfftDestroy(handle3);
    //---------------------------------------------------------------

    // Write both selections of data to unique audio files
    // Surrounding data
    AudioFile<float> output;
    string outputFilePath = "result_audio2.wav"; // change this to somewhere useful for you
    cout << outputFilePath << endl;

    for (int j = 0; j < size; j++) {
        output.samples[0].push_back(hostOutputData2[j] / size);
        output.samples[0].push_back(0);
    }
    output.save(outputFilePath, AudioFileFormat::Wave);
        cout << audio.size() << endl;

    // Sliced data
    AudioFile<float> output2;
    string outputFilePath2 = "result_audio3.wav"; // change this to somewhere useful for you
    cout << outputFilePath2 << endl;

    for (int j = 0; j < size; j++) {
        output2.samples[0].push_back(hostOutputData3[j] / size);
        output2.samples[0].push_back(0);
    }
    output2.save(outputFilePath2, AudioFileFormat::Wave);
    cout << audio.size() << endl;
}
