#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

using namespace std;
#include "AudioFile.h"

#define PROJECT_BINARY_DIR "C:/Users/mlaramie24/source/repos/AudioWork"

AudioFile<float> loadAudio(vector<float>& v) {
    //---------------------------------------------------------------
    // 1. Set a file path to an audio file on your machine

    const std::string inputFilePath = std::string(PROJECT_BINARY_DIR) + "/AudioWork/examples_test-audio.wav";

    //---------------------------------------------------------------
    // 2. Create an AudioFile object and load the audio file

    AudioFile<float> a;
    bool loadedOK = a.load(inputFilePath);

    //---------------------------------------------------------------
    // 3. Let's apply a gain to every audio sample

    for (int i = 0; i < a.getNumSamplesPerChannel(); i++) {
        for (int channel = 0; channel < a.getNumChannels(); channel++) {
            v.push_back(a.samples[channel][i]);
        }
    }
    return a;
}

int main() {
    std::cout << "**********************" << std::endl;
    std::cout << "funking arounddd" << std::endl;
    std::cout << "**********************" << std::endl << std::endl;


    //---------------------------------------------------------------
    // Step 1: Input Data
    vector<float> audio;
    AudioFile<float> a = loadAudio(audio);
    int bigness = audio.size();

    hipfftReal* hostInputData = (hipfftReal*)malloc(bigness * sizeof(hipfftReal));
    for (int i = 0; i < bigness; i++) {
        hostInputData[i] = (hipfftReal)audio[i];
    }

    //---------------------------------------------------------------
    // Step 2: Device memory allocation
    hipfftComplex* deviceInputData;
    hipMalloc((void**)&deviceInputData, bigness * sizeof(hipfftComplex));
    hipMemcpy(deviceInputData, hostInputData, bigness * sizeof(hipfftReal), hipMemcpyHostToDevice);

    //---------------------------------------------------------------
    // Step 3: Allocate host & device memory for output data collection
    hipfftComplex* hostOutputData = (hipfftComplex*)malloc((bigness / 2 + 1) * 1 * sizeof(hipfftComplex));

    hipfftComplex* deviceOutputData;
    hipMalloc((void**)&deviceOutputData, (bigness / 2 + 1) * sizeof(hipfftComplex));
    
    // Setup cufft plan and handle
    hipfftHandle handle;
    hipfftResult cufftStatus = hipfftPlan1d(&handle, bigness, HIPFFT_R2C, 1);

    //---------------------------------------------------------------
    // Step 4: Execute R2C FFT
    cufftStatus = hipfftExecR2C(handle, (hipfftReal*)deviceInputData, deviceOutputData);

    //---------------------------------------------------------------
    // Step 5: Transfer results from Device -> Host
    hipMemcpy(hostOutputData, deviceOutputData, (bigness / 2 + 1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < (bigness / 2 + 1); i++) {
        printf("%i %f %f\n", i, hostOutputData[i].x, hostOutputData[i].y);
    }

    // Cleanup
    hipfftDestroy(handle);

    //std::string outputFilePath = "result_audio.wav"; // change this to somewhere useful for you
    //a.save(outputFilePath, AudioFileFormat::Aiff);
    //cout << audio.size() << endl;
}